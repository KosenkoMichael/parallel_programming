#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <chrono>
#include <cstdlib>
#include <fstream>
#include <string>
#include <sstream>
#include <hip/hip_runtime.h>
#include <direct.h>

#define CUDA_CHECK(err) if (err != hipSuccess) { \
    std::cerr << "CUDA error: " << hipGetErrorString(err) << "\n"; \
    exit(1); \
}

void createDirectory(const std::string& path) {
    std::string command = "mkdir \"" + path + "\"";
    std::system(command.c_str());
}

std::vector<std::vector<int>> generateMatrix(int rows, int cols, int minVal, int maxVal) {
    if (minVal > maxVal) std::swap(minVal, maxVal);
    std::vector<std::vector<int>> matrix(rows, std::vector<int>(cols));
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dist(minVal, maxVal);

    for (int i = 0; i < rows; ++i)
        for (int j = 0; j < cols; ++j)
            matrix[i][j] = dist(gen);

    return matrix;
}

void writeMatrixToFile(const std::string& filename, const std::vector<std::vector<int>>& matrix) {
    std::ofstream out(filename);
    for (const auto& row : matrix) {
        for (int val : row)
            out << val << " ";
        out << "\n";
    }
}

__global__ void matrixMultiplyKernel(int* A, int* B, int* C, int N) {
    extern __shared__ int sharedMem[];

    int* sharedA = sharedMem;
    int* sharedB = &sharedMem[blockDim.x * blockDim.y];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;

    for (int t = 0; t < (N + blockDim.x - 1) / blockDim.x; ++t) {
        int indexA = threadIdx.y * blockDim.x + threadIdx.x;
        int indexB = threadIdx.y * blockDim.y + threadIdx.x;

        if (row < N && t * blockDim.x + threadIdx.x < N)
            sharedA[indexA] = A[row * N + t * blockDim.x + threadIdx.x];
        else
            sharedA[indexA] = 0;

        if (col < N && t * blockDim.y + threadIdx.y < N)
            sharedB[indexB] = B[(t * blockDim.y + threadIdx.y) * N + col];
        else
            sharedB[indexB] = 0;

        __syncthreads();

        for (int k = 0; k < blockDim.x; ++k)
            sum += sharedA[threadIdx.y * blockDim.x + k] * sharedB[k * blockDim.y + threadIdx.x];

        __syncthreads();
    }

    if (row < N && col < N)
        C[row * N + col] = sum;
}

int main(int argc, char* argv[]) {
    if (argc < 7) {
        std::cerr << "Usage: ./cuda_matrix_mul <N> <minVal> <maxVal> <subfolder> <threadsX> <threadsY>\n";
        return 1;
    }

    int N = std::atoi(argv[1]);
    int minVal = std::atoi(argv[2]);
    int maxVal = std::atoi(argv[3]);
    std::string subpath = argv[4];
    int threadsX = std::atoi(argv[5]);
    int threadsY = std::atoi(argv[6]);

    if (threadsX <= 0 || threadsY <= 0 || threadsX * threadsY > 1024) {
        std::cerr << "Error: threadsX * threadsY must be > 0 and <= 1024\n";
        return 1;
    }

    auto A = generateMatrix(N, N, minVal, maxVal);
    auto B = generateMatrix(N, N, minVal, maxVal);

    std::vector<int> A_flat(N * N), B_flat(N * N), C_flat(N * N);
    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j) {
            A_flat[i * N + j] = A[i][j];
            B_flat[i * N + j] = B[i][j];
        }

    int* d_A;
    int* d_B;
    int* d_C;
    size_t size = N * N * sizeof(int);

    CUDA_CHECK(hipMalloc(&d_A, size));
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_C, size));

    CUDA_CHECK(hipMemcpy(d_A, A_flat.data(), size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B_flat.data(), size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(threadsX, threadsY);
    dim3 blocksPerGrid((N + threadsX - 1) / threadsX, (N + threadsY - 1) / threadsY);

    size_t sharedMemSize = 2 * threadsX * threadsY * sizeof(int);

    auto start = std::chrono::high_resolution_clock::now();
    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();

    long long elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

    CUDA_CHECK(hipMemcpy(C_flat.data(), d_C, size, hipMemcpyDeviceToHost));

    std::vector<std::vector<int>> C(N, std::vector<int>(N));
    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            C[i][j] = C_flat[i * N + j];

    std::string folderName = "result\\cuda_" + std::to_string(N) + subpath;
    createDirectory("result");
    createDirectory(folderName);

    writeMatrixToFile(folderName + "\\A.txt", A);
    writeMatrixToFile(folderName + "\\B.txt", B);
    writeMatrixToFile(folderName + "\\C.txt", C);

    std::cout << "CUDA Matrix Multiplication Completed\n";
    std::cout << "Matrix size: " << N << "x" << N << " Time: " << elapsedTime << " microseconds\n";

    bool isNewFile = false;
    std::ifstream test("stats.csv");
    if (!test.good() || test.peek() == std::ifstream::traits_type::eof())
        isNewFile = true;
    test.close();

    std::ofstream stats("stats.csv", std::ios::app);
    if (isNewFile)
        stats << "ThreadsX,ThreadsY,MatrixSize,MinValue,MaxValue,Time(mcs)\n";

    stats << threadsX << "," << threadsY << "," << N << "," << minVal << "," << maxVal << "," << elapsedTime << "\n";
    stats.close();

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    return 0;
}
